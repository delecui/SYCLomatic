#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none --usm-level=none --in-root=%S --out-root=%T/out --analysis-scope-path=%S/.. %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// out/
// ├── kernel_warp.dp.cpp
// └── MainSourceFiles.yaml
// RUN: echo > %T/exist_check
// RUN: bash %S/../check_script.sh %T/out/kernel_warp.dp.cpp %T
// RUN: bash %S/../check_script.sh %T/out/MainSourceFiles.yaml %T
// RUN: bash %S/../check_script.sh %T/out/inc/empty.h %T
// RUN: bash %S/../check_script.sh %T/out/inc/utils.dp.hpp %T
// RUN: bash %S/../check_script.sh %T/out/src %T
// RUN: FileCheck --input-file %T/exist_check --match-full-lines %S/../ref
// RUN: rm -rf %T/out
#include "../inc/utils.cuh"
#include "../inc/empty.h"

//CHECK:void kernel(float *input, sycl::nd_item<3> item_ct1, float *smem) {
__global__ void kernel(float *input) {
  float sum = 0;
  __shared__ float smem[128];
  //CHECK:float total_sum = BlockReduceSum(sum, smem, item_ct1);
  float total_sum = BlockReduceSum(sum, smem);
}

void foo() {
  float *input = NULL;
  //CHECK:dpct::get_default_queue().submit(
  //CHECK-NEXT:  [&](sycl::handler &cgh) {
  //CHECK-NEXT:    sycl::accessor<float, 1, sycl::access_mode::read_write, sycl::access::target::local> smem_acc_ct1(sycl::range<1>(128), cgh);
  //CHECK-NEXT:    dpct::access_wrapper<float *> input_acc_ct0(input, cgh);
  //CHECK-EMPTY:
  //CHECK-NEXT:    cgh.parallel_for(
  //CHECK-NEXT:      sycl::nd_range<3>(sycl::range<3>(1, 1, 128), sycl::range<3>(1, 1, 128)), 
  //CHECK-NEXT:      [=](sycl::nd_item<3> item_ct1) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  //CHECK-NEXT:        kernel(input_acc_ct0.get_raw_pointer(), item_ct1, smem_acc_ct1.get_pointer());
  //CHECK-NEXT:      });
  //CHECK-NEXT:  });
  kernel<<<1, 128>>>(input);
}